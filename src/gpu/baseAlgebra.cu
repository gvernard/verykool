#include "hip/hip_runtime.h"
#include "baseAlgebra.hpp"

#include "sourcePlane.hpp"

#include "cusparse_cholesky_solver.h"

using Ordering = Eigen::AMDOrdering<Eigen::SparseMatrix<double,Eigen::StorageOptions::RowMajor>::StorageIndex>;
using PermutationMatrix = Ordering::PermutationType;

// Class: BaseAlgebra
//===============================================================================================================
void BaseAlgebra::setAlgebraField(BaseSourcePlane* source,Eigen::SparseMatrix<double> mat_in,Eigen::SparseMatrix<double>& mat_out,double& det_out){
  Eigen::SparseMatrix<double> out(source->Sm,source->Sm);

  if( source->reg == "covariance_kernel" || source->reg == "identity" ){
    this->getInverseAndDet(mat_in,out,det_out);
  } else {
    // calculate the product HtH of a derivative based H matrix
    Eigen::SparseMatrix<double> mat_in_t(mat_in.rows(),mat_in.cols());
    mat_in_t = mat_in.transpose();
    out = (mat_in_t * mat_in);    
    mat_in_t.resize(0,0);
    det_out = -this->getDeterminant(out); // the minus sign is because Cs is actually (HtH)^-1
  }
  
  mat_out = out;
  out.resize(0,0);
}

void BaseAlgebra::getInverseAndDet(Eigen::SparseMatrix<double> mat_in,Eigen::SparseMatrix<double>& mat_out,double& det_out){
  Eigen::SparseMatrix<double,Eigen::StorageOptions::RowMajor> Acsr = mat_in; // solver supports CSR format
  Eigen::Matrix<double,Eigen::Dynamic,Eigen::Dynamic> inv(mat_in.rows(),mat_in.cols());
  Eigen::VectorXd idc(mat_in.cols()),c(mat_in.cols());
  int nnz = mat_in.nonZeros();
  double det = 0.0;


  auto solver = CuSparseCholeskySolver<double>::create(mat_in.cols());
  
  
  // compute permutation
  PermutationMatrix P;
  Ordering ordering;
  ordering(Acsr.selfadjointView<Eigen::Upper>(), P);
  // set permutation to solver
  solver->setPermutaion(mat_in.cols(), P.indices().data());



  solver->analyze(nnz, Acsr.outerIndexPtr(), Acsr.innerIndexPtr());
  solver->factorize(Acsr.valuePtr());

  if( solver->info() != CuSparseCholeskySolver<double>::SUCCESS ){
    std::cerr << "Factorize failed." << std::endl;
    std::exit(EXIT_FAILURE);
  }

  std::cout << "Factorization done" << std::endl;


  for(int i=0;i<mat_in.rows();i++){
    for(int j=0;j<idc.size();j++){
      idc[j] = 0;
    }
    idc[i] = 1;
    //this needs to be done in two steps, otherwise it does not work when using LU
    solver->solve(idc.data(),c.data());
    inv.col(i) = c;
  }



  /*
  // get determinant of mat_in (NOT its inverse)
  Eigen::VectorXd diag = solver.vectorD();
  diag = solver.vectorD();
  for(int i=0;i<diag.size();i++){
    det += log( *(diag.data()+i) );
    //    dum = *(diag.data()+i);
    //    if( dum > 1.e-20 ){
    //      det += log( dum );
    //    }
  }
  */
  det = 0.0;

  
  det_out = det;
  mat_out = inv.sparseView();
  inv.resize(0,0);
}

double BaseAlgebra::getDeterminant(Eigen::SparseMatrix<double> mat){
  Eigen::SimplicialLDLT< Eigen::SparseMatrix<double> > solver;
  Eigen::VectorXd diag;
  double dum;
  
  double det = 0.0;
  /*
  solver.analyzePattern(mat);
  solver.factorize(mat);
  diag = solver.vectorD();
  for(int i=0;i<diag.size();i++){
    //    std::cout << " " << *(diag.data()+i);
    det += log( *(diag.data()+i) );
    //    dum = *(diag.data()+i);
    //    if( dum > 1.e-20 ){
    //      det += log( dum );
    //    }
  }
  //  std::cout << std::endl << std::endl;
  */
  return det;
}

